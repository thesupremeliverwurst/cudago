#include "hip/hip_runtime.h"
extern "C"{
	#include "Kernalize.h"
}
// typedef void (*gFunc) ();
// __global__
// void Kernalize::kernalize(gFunc *ptr) {
//     (*ptr)();
// }
// void Kernalize::start(gFunc *ptr) {
// 	kernalize<<<1,1>>>(ptr);
// }
extern "C" {
	typedef void (*gFunc) () __asm__ ("cudago.tobe");
	__global__ void kernalize(gFunc* ptr) {
	    (*ptr)();
	}
	void Start(gFunc* ptr) {
		kernalize<<<1,1>>>(ptr);
	}
}